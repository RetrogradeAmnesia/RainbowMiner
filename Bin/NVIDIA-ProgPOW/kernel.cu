#include "hip/hip_runtime.h"
typedef unsigned int       uint32_t;
typedef unsigned long long uint64_t;
#define ROTL32(x,n) __funnelshift_l((x), (x), (n))
#define ROTR32(x,n) __funnelshift_r((x), (x), (n))
#define min(a,b) ((a<b) ? a : b)
#define mul_hi(a, b) __umulhi(a, b)
#define clz(a) __clz(a)
#define popcount(a) __popc(a)

#define PROGPOW_LANES			32
#define PROGPOW_REGS			16
#define PROGPOW_CNT_MEM			64
#define PROGPOW_CNT_MATH		8
#define PROGPOW_CACHE_WORDS  4096

__device__ __forceinline__ void progPowLoop(const uint32_t loop,
        uint32_t mix[PROGPOW_REGS],
        const uint64_t *g_dag,
        const uint32_t c_dag[PROGPOW_CACHE_WORDS])
{
uint32_t offset;
uint64_t data64;
uint32_t data32;
const uint32_t lane_id = threadIdx.x & (PROGPOW_LANES-1);
// global load
offset = __shfl_sync(0xFFFFFFFF, mix[0], loop%PROGPOW_LANES, PROGPOW_LANES);
offset %= PROGPOW_DAG_WORDS;
offset = offset * PROGPOW_LANES + lane_id;
data64 = g_dag[offset];
// cache load
offset = mix[4] % PROGPOW_CACHE_WORDS;
data32 = c_dag[offset];
mix[4] = ROTL32(mix[4], 3) ^ data32;
// random math
data32 = ROTR32(mix[13], mix[9]);
mix[15] = ROTR32(mix[15], 31) ^ data32;
// cache load
offset = mix[8] % PROGPOW_CACHE_WORDS;
data32 = c_dag[offset];
mix[13] = ROTR32(mix[13], 21) ^ data32;
// random math
data32 = min(mix[2], mix[12]);
mix[10] = (mix[10] * 33) + data32;
// cache load
offset = mix[7] % PROGPOW_CACHE_WORDS;
data32 = c_dag[offset];
mix[11] = (mix[11] * 33) + data32;
// random math
data32 = mul_hi(mix[1], mix[1]);
mix[3] = ROTL32(mix[3], 28) ^ data32;
// cache load
offset = mix[15] % PROGPOW_CACHE_WORDS;
data32 = c_dag[offset];
mix[7] = ROTR32(mix[7], 12) ^ data32;
// random math
data32 = mix[9] | mix[7];
mix[9] = (mix[9] ^ data32) * 33;
// cache load
offset = mix[10] % PROGPOW_CACHE_WORDS;
data32 = c_dag[offset];
mix[12] = (mix[12] * 33) + data32;
// random math
data32 = mix[6] * mix[14];
mix[2] = ROTL32(mix[2], 3) ^ data32;
// cache load
offset = mix[14] % PROGPOW_CACHE_WORDS;
data32 = c_dag[offset];
mix[1] = (mix[1] * 33) + data32;
// random math
data32 = mul_hi(mix[3], mix[1]);
mix[14] = ROTL32(mix[14], 7) ^ data32;
// cache load
offset = mix[15] % PROGPOW_CACHE_WORDS;
data32 = c_dag[offset];
mix[6] = (mix[6] ^ data32) * 33;
// random math
data32 = clz(mix[0]) + clz(mix[8]);
mix[5] = (mix[5] ^ data32) * 33;
// cache load
offset = mix[15] % PROGPOW_CACHE_WORDS;
data32 = c_dag[offset];
mix[0] = (mix[0] * 33) + data32;
// random math
data32 = mix[10] | mix[1];
mix[8] = ROTL32(mix[8], 28) ^ data32;
// consume global load data
mix[0] = (mix[0] ^ data64) * 33;
mix[4] = (mix[4] * 33) + (data64>>32);
}

#ifndef SEARCH_RESULTS
#define SEARCH_RESULTS 4
#endif

#define FNV_PRIME ((uint32_t)0x1000193)

typedef struct {
    uint32_t count;
    struct {
        // One word for gid and 8 for mix hash
        uint32_t gid;
        uint32_t mix[8];
    } result[SEARCH_RESULTS];
} search_results;

typedef struct
{
    uint32_t uint32s[32 / sizeof(uint32_t)];
} hash32_t;

// Implementation based on:
// https://github.com/mjosaarinen/tiny_sha3/blob/master/sha3.c

__device__ __constant__ const uint32_t keccakf_rndc[24] = {
        0x00000001, 0x00008082, 0x0000808a, 0x80008000, 0x0000808b, 0x80000001,
        0x80008081, 0x00008009, 0x0000008a, 0x00000088, 0x80008009, 0x8000000a,
        0x8000808b, 0x0000008b, 0x00008089, 0x00008003, 0x00008002, 0x00000080,
        0x0000800a, 0x8000000a, 0x80008081, 0x00008080, 0x80000001, 0x80008008
};

// Implementation of the permutation Keccakf with width 800.
__device__ __forceinline__ void keccak_f800_round(uint32_t st[25], const int r)
{

    const uint32_t keccakf_rotc[24] = {
            1,  3,  6,  10, 15, 21, 28, 36, 45, 55, 2,  14,
            27, 41, 56, 8,  25, 43, 62, 18, 39, 61, 20, 44
    };
    const int keccakf_piln[24] = {
            10, 7,  11, 17, 18, 3, 5,  16, 8,  21, 24, 4,
            15, 23, 19, 13, 12, 2, 20, 14, 22, 9,  6,  1
    };

    uint32_t t, bc[5];
    // Theta
    for (int i = 0; i < 5; i++)
        bc[i] = st[i] ^ st[i + 5] ^ st[i + 10] ^ st[i + 15] ^ st[i + 20];

    for (int i = 0; i < 5; i++) {
        t = bc[(i + 4) % 5] ^ ROTL32(bc[(i + 1) % 5], 1u);
        for (int j = 0; j < 25; j += 5)
            st[j + i] ^= t;
    }

    // Rho Pi
    t = st[1];
    for (int i = 0; i < 24; i++) {
        uint32_t j = keccakf_piln[i];
        bc[0] = st[j];
        st[j] = ROTL32(t, keccakf_rotc[i]);
        t = bc[0];
    }

    //  Chi
    for (int j = 0; j < 25; j += 5) {
        for (int i = 0; i < 5; i++)
            bc[i] = st[j + i];
        for (int i = 0; i < 5; i++)
            st[j + i] ^= (~bc[(i + 1) % 5]) & bc[(i + 2) % 5];
    }

    //  Iota
    st[0] ^= keccakf_rndc[r];
}

// Implementation of the Keccak sponge construction (with padding omitted)
// The width is 800, with a bitrate of 576, and a capacity of 224.
__device__ __noinline__ uint64_t keccak_f800(hash32_t header, uint64_t seed, hash32_t result)
{
    uint32_t st[25];

#pragma unroll
    for (int i = 0; i < 25; i++)
        st[i] = 0;

#pragma unroll
    for (int i = 0; i < 8; i++)
        st[i] = header.uint32s[i];

    st[8] = (uint32_t)seed;
    st[9] = (uint32_t)(seed >> 32);
#pragma unroll
    for (int i = 0; i < 8; i++)
        st[10+i] = result.uint32s[i];

    for (int r = 0; r < 21; r++) {
        keccak_f800_round(st, r);
    }
    // last round can be simplified due to partial output
    keccak_f800_round(st, 21);

    return (uint64_t)st[0] << 32 | (uint64_t)st[1];
}

__device__ __forceinline__ uint32_t fnv1a(uint32_t *h, uint32_t d)
{
    *h = (*h ^ d) * FNV_PRIME;
    return *h;
}

typedef struct {
    uint32_t z, w, jsr, jcong;
} kiss99_t;

// KISS99 is simple, fast, and passes the TestU01 suite
// https://en.wikipedia.org/wiki/KISS_(algorithm)
// http://www.cse.yorku.ca/~oz/marsaglia-rng.html
__device__ __forceinline__ uint32_t kiss99(kiss99_t *st)
{
    uint32_t znew = (st->z = 36969 * (st->z & 65535) + (st->z >> 16));
    uint32_t wnew = (st->w = 18000 * (st->w & 65535) + (st->w >> 16));
    uint32_t MWC = ((znew << 16) + wnew);
    uint32_t SHR3 = (st->jsr ^= (st->jsr << 17), st->jsr ^= (st->jsr >> 13), st->jsr ^= (st->jsr << 5));
    uint32_t CONG = (st->jcong = 69069 * st->jcong + 1234567);
    return ((MWC^CONG) + SHR3);
}

__device__ __forceinline__ void fill_mix(uint64_t seed, uint32_t lane_id, uint32_t mix[PROGPOW_REGS])
{
    // Use FNV to expand the per-warp seed to per-lane
    // Use KISS to expand the per-lane seed to fill mix
    uint32_t fnv_hash = 0x811c9dc5;
    kiss99_t st;
    st.z = fnv1a(&fnv_hash, (uint32_t)seed);
    st.w = fnv1a(&fnv_hash, (uint32_t)(seed >> 32));
    st.jsr = fnv1a(&fnv_hash, lane_id);
    st.jcong = fnv1a(&fnv_hash, lane_id);
#pragma unroll
    for (int i = 0; i < PROGPOW_REGS; i++)
        mix[i] = kiss99(&st);
}

__global__ void progpow_search(
        uint64_t start_nonce,
        const hash32_t header,
        const uint64_t target,
        const uint64_t *g_dag,
        volatile search_results* g_output
)
{
    __shared__ uint32_t c_dag[PROGPOW_CACHE_WORDS];
    uint32_t const gid = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t const nonce = start_nonce + gid;

    const uint32_t lane_id = threadIdx.x & (PROGPOW_LANES - 1);

    // Load random data into the cache
    // TODO: should be a new blob of data, not existing DAG data
    for (uint32_t word = threadIdx.x*2; word < PROGPOW_CACHE_WORDS; word += blockDim.x*2)
    {
        uint64_t data = g_dag[word];
        c_dag[word + 0] = data;
        c_dag[word + 1] = data >> 32;
    }

    hash32_t result;
#pragma unroll
    for (int i = 0; i < 8; i++)
        result.uint32s[i] = 0;
    // keccak(header..nonce)
    uint64_t seed = keccak_f800(header, nonce, result);

    __syncthreads();

#pragma unroll 1
    for (uint32_t h = 0; h < PROGPOW_LANES; h++)
    {
        uint32_t mix[PROGPOW_REGS];

        // share the hash's seed across all lanes
        uint64_t hash_seed = __shfl_sync(0xFFFFFFFF, seed, h, PROGPOW_LANES);
        // initialize mix for all lanes
        fill_mix(hash_seed, lane_id, mix);

#pragma unroll 1
        for (uint32_t l = 0; l < PROGPOW_CNT_MEM; l++)
            progPowLoop(l, mix, g_dag, c_dag);


        // Reduce mix data to a single per-lane result
        uint32_t result_lane = 0x811c9dc5;
#pragma unroll
        for (int i = 0; i < PROGPOW_REGS; i++)
            fnv1a(&result_lane, mix[i]);

        // Reduce all lanes to a single 256-bit result
        hash32_t result_hash;
#pragma unroll
        for (int i = 0; i < 8; i++)
            result_hash.uint32s[i] = 0x811c9dc5;

        for (int i = 0; i < PROGPOW_LANES; i += 8)
#pragma unroll
                for (int j = 0; j < 8; j++)
                    fnv1a(&result_hash.uint32s[j], __shfl_sync(0xFFFFFFFF, result_lane, i + j, PROGPOW_LANES));

        if (h == lane_id)
            result = result_hash;
    }

    // keccak(header .. keccak(header..nonce) .. result);
    if (keccak_f800(header, seed, result) > target)
        return;

    uint32_t index = atomicInc((uint32_t *)&g_output->count, 0xffffffff);
    if (index >= SEARCH_RESULTS)
        return;

    g_output->result[index].gid = gid;
#pragma unroll
    for (int i = 0; i < 8; i++)
        g_output->result[index].mix[i] = result.uint32s[i];
}